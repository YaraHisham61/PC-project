#include "hip/hip_runtime.h"
#include "kernels/filter_kernel.hpp"

template <typename T>
__global__ void filterKernel(const T *input, bool *output, size_t row_count, T value, uint8_t cond)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= row_count)
        return;

    switch (cond)
    {
    case 1: // >

        output[idx] = (input[idx] > value) ? true : false;
        break;
    case 2: // <
        output[idx] = (input[idx] < value) ? true : false;
        break;
    case 3: // ==
        output[idx] = (input[idx] == value) ? true : false;
        break;
    case 4: // !=
        output[idx] = (input[idx] != value) ? true : false;
        break;
    case 5: // <=
        output[idx] = (input[idx] <= value) ? true : false;
        break;
    case 6: // >=
        output[idx] = (input[idx] >= value) ? true : false;
        break;
    default:
        output[idx] = false;
        break;
    }
}

__device__ int device_strcmp(const char *s1, const char *s2)
{
    while (*s1 && (*s1 == *s2))
    {
        s1++;
        s2++;
    }
    return *(const unsigned char *)s1 - *(const unsigned char *)s2;
}

__global__ void filterKernelString(const char **input, bool *output, size_t row_count, const char *value, uint8_t cond)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= row_count)
        return;

    output[idx] = (device_strcmp(input[idx], value) == 0);
}

__global__ void andKernel(bool *combined_mask, const bool *current_mask, size_t size)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        combined_mask[idx] = combined_mask[idx] && current_mask[idx];
    }
}

__global__ void orKernel(bool *combined_mask, const bool *current_mask, size_t size)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        combined_mask[idx] = combined_mask[idx] || current_mask[idx];
    }
}

// template <typename T>
// __global__ void filterColumnKernel(const T *input, T *output, const bool *mask,
//                                    const size_t row_count, unsigned long long *filtered_count)
// {
//     size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= row_count)
//         return;

//     if (mask[idx])
//     {
//         size_t pos = atomicAdd(filtered_count, 1ull);
//         if (pos < row_count)
//         { // Safety check
//             output[pos] = input[idx];
//         }
//     }
// }

// __global__ void filterStringColumnKernel(const char **input, char **output, const bool *mask, const size_t row_count, unsigned long long *filtered_count)
// {
//     size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= row_count)
//         return;
//     if (mask[idx])
//     {
//         printf("idx: %zu, mask[idx]: %d\n", idx, mask[idx]);
//         printf("input[idx]: %s\n", input[idx]);
//         printf("output: %p\n", output);
//         printf("filtered_count: %llu\n", *filtered_count);
//         size_t pos = atomicAdd(filtered_count, 1ull);
//         if (pos < row_count)
//         {
//             output[pos] = const_cast<char *>(input[idx]);
//         }
//     }
// }

__global__ void computeOutputPositions(const bool *mask, unsigned int *positions, size_t size)
{
    extern __shared__ unsigned int temp[];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    // Load and calculate block prefix
    unsigned int val = (idx < size && mask[idx]) ? 1 : 0;
    temp[tid] = val;
    __syncthreads();

    // Parallel prefix sum within block
    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid >= s)
        {
            temp[tid] += temp[tid - s];
        }
        __syncthreads();
    }

    // Store block result
    if (idx < size)
    {
        positions[idx] = temp[tid];
    }
    __syncthreads();

    // If last thread in block, store block total
    if (tid == blockDim.x - 1)
    {
        temp[0] = temp[tid]; // Store block sum
    }
    __syncthreads();

    // First block adds prefix sums from previous blocks
    if (blockIdx.x > 0)
    {
        if (tid == 0)
        {
            // Get sum from previous block
            unsigned int prefix = 0;
            for (int i = 0; i < blockIdx.x; i++)
            {
                unsigned int *prev_block = positions + i * blockDim.x;
                prefix += prev_block[blockDim.x - 1];
            }
            // Add prefix to all elements in this block
            for (int i = 0; i < blockDim.x && (blockIdx.x * blockDim.x + i) < size; i++)
            {
                positions[blockIdx.x * blockDim.x + i] += prefix;
            }
        }
        __syncthreads();
    }
}

template <typename T>
__global__ void copySelectedRowsKernel(const T *input, T *output,
                                       const bool *mask, const unsigned int *positions,
                                       size_t size)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && mask[idx])
    {
        output[positions[idx] - 1] = input[idx];
    }
}

__global__ void copySelectedStringRowsKernel(const char **input, const char **output,
                                             const bool *mask, const unsigned int *positions,
                                             size_t size)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && mask[idx])
    {
        output[positions[idx] - 1] = input[idx];
    }
}

template __global__ void filterKernel<int>(const int *, bool *, size_t, int, uint8_t);
template __global__ void filterKernel<float>(const float *, bool *, size_t, float, uint8_t);
template __global__ void filterKernel<int64_t>(const int64_t *, bool *, size_t, int64_t, uint8_t);

template __global__ void copySelectedRowsKernel<int>(const int *, int *, const bool *, const unsigned int *, const size_t);
template __global__ void copySelectedRowsKernel<float>(const float *, float *, const bool *, const unsigned int *, const size_t);
template __global__ void copySelectedRowsKernel<int64_t>(const int64_t *, int64_t *, const bool *, const unsigned int *, const size_t);
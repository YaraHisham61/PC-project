#include "hip/hip_runtime.h"
#include "kernels/order_by.hpp"

template <typename T>
__device__ bool operators(const T &a, const T &b, bool ascending)
{
    return ascending ? (a < b) : (a > b);
}

template <>
__device__ bool operators<char *>(char *const &a, char *const &b, bool ascending)
{
    int cmp = 0;
    int i = 0;
    while (a[i] != '\0' && b[i] != '\0')
    {
        if (a[i] != b[i])
        {
            cmp = a[i] - b[i];
            break;
        }
        i++;
    }
    if (cmp == 0)
    {
        if (a[i] == '\0' && b[i] != '\0')
            cmp = -1;
        else if (a[i] != '\0' && b[i] == '\0')
            cmp = 1;
    }
    return ascending ? (cmp < 0) : (cmp > 0);
}

template <typename T>
__device__ void merge(T *keys, size_t *indices, size_t *indicesTmp,
                      int left, int mid, int right, bool ascending)
{
    int i = left;   
    int j = mid + 1; 
    int k = left;   

    while (i <= mid && j <= right)
    {
        if (operators<T>(keys[indices[i]], keys[indices[j]], ascending))
        {
            indicesTmp[k] = indices[i];
            i++;
        }
        else
        {
            indicesTmp[k] = indices[j];
            j++;
        }
        k++;
    }

    while (i <= mid)
    {
        indicesTmp[k] = indices[i];
        i++;
        k++;
    }

    while (j <= right)
    {
        indicesTmp[k] = indices[j];
        j++;
        k++;
    }

    for (i = left; i <= right; i++)
    {
        indices[i] = indicesTmp[i];
    }
}

template <>
__device__ void merge<char *>(char **keys, size_t *indices, size_t *indicesTmp,
                              int left, int mid, int right, bool ascending)
{
    int i = left;    
    int j = mid + 1; 
    int k = left;    

    while (i <= mid && j <= right)
    {
        if (operators<char *>(keys[indices[i]], keys[indices[j]], ascending))
        {
            indicesTmp[k] = indices[i];
            i++;
        }
        else
        {
            indicesTmp[k] = indices[j];
            j++;
        }
        k++;
    }

    while (i <= mid)
    {
        indicesTmp[k] = indices[i];
        i++;
        k++;
    }

    while (j <= right)
    {
        indicesTmp[k] = indices[j];
        j++;
        k++;
    }

    for (i = left; i <= right; i++)
    {
        indices[i] = indicesTmp[i];
    }
}

template <typename T>
__global__ void mergeSortKernel(T *keys, size_t *indices, size_t *indicesTmp,
                                int n, int width, bool ascending)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int left = tid * 2 * width;

    if (left >= n)
        return;

    int mid = min(left + width - 1, n - 1);
    int right = min(left + 2 * width - 1, n - 1);

    merge<T>(keys, indices, indicesTmp, left, mid, right, ascending);
}

template __global__ void mergeSortKernel<float>(
    float *, size_t *, size_t *,
    int, int, bool);

template __global__ void mergeSortKernel<uint64_t>(
    uint64_t *, size_t *, size_t *,
    int, int, bool);

    
template __global__ void mergeSortKernel<char *>(
    char **, size_t *, size_t *,
    int, int, bool);

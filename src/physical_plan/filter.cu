#include "hip/hip_runtime.h"
#include "physical_plan/filter.hpp"

Filter::Filter(const duckdb::InsertionOrderPreservingMap<std::string> &params)
    : PhysicalOpNode()
{
    auto it = params.find("__expression__");
    if (it != params.end())
    {
        parseConditions(it->second);
        removeTimestampSuffixSimple();
    }
}
void Filter::removeTimestampSuffixSimple()
{
    const std::string suffix = "::TIMESTAMP";

    for (auto &cond : conditions)
    {
        size_t pos = cond.value.rfind(suffix);
        if (pos != std::string::npos)
        {
            cond.value = cond.value.substr(1, pos - 1);
        }
    }
}
std::string Filter::trim(const std::string &str) const
{
    size_t first = str.find_first_not_of(" \t\n\r");
    if (first == std::string::npos)
        return "";
    size_t last = str.find_last_not_of(" \t\n\r");
    return str.substr(first, (last - first + 1));
}

void Filter::parseConditions(const std::string &expression)
{
    std::string expr = trim(expression);

    size_t and_pos = expr.find(" AND ");
    size_t or_pos = expr.find(" OR ");

    if (and_pos != std::string::npos || or_pos != std::string::npos)
    {
        parseComplexCondition(expr);
    }
    else
    {
        parseSimpleCondition(expr);
    }
}

void Filter::parseSimpleCondition(const std::string &expr)
{
    std::string inner = trim(expr);
    if (inner.front() == '(' && inner.back() == ')')
    {
        inner = inner.substr(1, inner.length() - 2);
    }

    Condition cond = parseSingleCondition(inner);
    if (!cond.column.empty())
    {
        conditions.push_back(cond);
    }
}

void Filter::parseComplexCondition(const std::string &expr)
{
    std::string inner = trim(expr);
    if (inner.front() == '(' && inner.back() == ')')
    {
        inner = inner.substr(1, inner.length() - 2);
    }

    std::vector<std::string> tokens;
    std::string current;
    int paren_level = 0;

    for (char c : inner)
    {
        if (c == '(')
            paren_level++;
        if (c == ')')
            paren_level--;

        if (paren_level == 0 && c == ' ')
        {
            if (!current.empty())
            {
                tokens.push_back(current);
                current.clear();
            }
        }
        else
        {
            current += c;
        }
    }
    if (!current.empty())
        tokens.push_back(current);

    for (size_t i = 0; i < tokens.size();)
    {
        if (tokens[i] == "AND" || tokens[i] == "OR")
        {
            logical_ops.push_back(tokens[i]);
            i++;
        }
        else
        {
            parseSimpleCondition(tokens[i]);
            i++;
        }
    }
}

Condition Filter::parseSingleCondition(const std::string &cond_expr) const
{
    Condition cond;
    std::regex op_regex(R"((>=|<=|!=|>|<|=))");
    std::smatch op_match;

    if (std::regex_search(cond_expr, op_match, op_regex))
    {
        cond.column = trim(cond_expr.substr(0, op_match.position()));
        cond.op = op_match.str();
        cond.value = trim(cond_expr.substr(op_match.position() + cond.op.length()));

        if (!cond.value.empty() && cond.value.front() == '\'' && cond.value.back() == '\'')
        {
            cond.value = cond.value.substr(1, cond.value.length() - 2);
            cond.is_string = true;
        }
        else
        {
            cond.is_string = false;
        }
    }
    return cond;
}

bool *Filter::getSelectedRows(const TableResults &input_table) const
{
    const size_t row_count = input_table.row_count;
    bool *h_final_mask = new bool[row_count]();

    bool *d_combined_mask = nullptr;
    hipMalloc(&d_combined_mask, row_count * sizeof(bool));
    hipMemset(d_combined_mask, 0, row_count * sizeof(bool));

    int numThreads = 256;
    int numBlocks = (row_count + numThreads - 1) / numThreads;

    for (size_t cond_idx = 0; cond_idx < this->conditions.size(); cond_idx++)
    {
        const auto &cond = this->conditions[cond_idx];
        const size_t col_idx = input_table.getColumnIndex(cond.column);
        const DataType col_type = input_table.columns[col_idx].type;

        uint8_t cond_code = 0;
        if (cond.op == ">")
            cond_code = 1;
        else if (cond.op == "<")
            cond_code = 2;
        else if (cond.op == "=")
            cond_code = 3;
        else if (cond.op == "!=")
            cond_code = 4;
        else if (cond.op == "<=")
            cond_code = 5;
        else if (cond.op == ">=")
            cond_code = 6;
        else
        {
            std::cerr << "Unsupported operator: " << cond.op << "\n";
            continue;
        }

        bool *d_temp_mask = nullptr;
        hipMalloc(&d_temp_mask, row_count * sizeof(bool));
        hipMemset(d_temp_mask, 0, row_count * sizeof(bool));

        if (col_type == DataType::FLOAT)
        {
            float *d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(float));
            hipMemcpy(d_col_data, input_table.data[col_idx], row_count * sizeof(float), hipMemcpyHostToDevice);
            float value = std::stof(cond.value);
            filterKernel<float><<<numBlocks, numThreads>>>(d_col_data, d_temp_mask, row_count, value, cond_code);
            hipFree(d_col_data);
        }
        else if (col_type == DataType::DATETIME)
        {
            uint64_t *d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(uint64_t));
            hipMemcpy(d_col_data, input_table.data[col_idx], row_count * sizeof(uint64_t), hipMemcpyHostToDevice);
            uint64_t value = getDateTime(cond.value);
            filterKernel<uint64_t><<<numBlocks, numThreads>>>(d_col_data, d_temp_mask, row_count, value, cond_code);
            hipFree(d_col_data);
        }
        else if (col_type == DataType::STRING)
        {
            const char **d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(char *));
            const char **host_strings = static_cast<const char **>(input_table.data[col_idx]);

            char **d_strings = new char *[row_count];

            for (size_t i = 0; i < row_count; i++)
            {
                size_t len = strlen(host_strings[i]) + 1;
                hipMalloc(&d_strings[i], len);
                hipMemcpy(d_strings[i], host_strings[i], len, hipMemcpyHostToDevice);
                hipMemcpy(&d_col_data[i], &d_strings[i], sizeof(char *), hipMemcpyHostToDevice);
            }

            char *d_value = nullptr;
            hipMalloc(&d_value, cond.value.size() + 1);
            hipMemcpy(d_value, cond.value.c_str(), cond.value.size() + 1, hipMemcpyHostToDevice);
            filterKernelString<<<numBlocks, numThreads>>>(d_col_data, d_temp_mask, row_count, d_value, cond_code);

            for (size_t i = 0; i < row_count; i++)
            {
                hipFree(d_strings[i]);
            }
            delete[] d_strings;
            hipFree(d_value);
            hipFree(d_col_data);
        }

        if (cond_idx == 0)
        {
            hipMemcpy(d_combined_mask, d_temp_mask, row_count * sizeof(bool), hipMemcpyDeviceToDevice);
        }
        else
        {
            const std::string &op = logical_ops[cond_idx - 1];
            if (op == "AND")
            {
                andKernel<<<numBlocks, numThreads>>>(d_combined_mask, d_temp_mask, row_count);
            }
            else if (op == "OR")
            {
                orKernel<<<numBlocks, numThreads>>>(d_combined_mask, d_temp_mask, row_count);
            }
        }
        bool *h_temp_mask = new bool[row_count];
        hipMemcpy(h_temp_mask, d_temp_mask, row_count * sizeof(bool), hipMemcpyDeviceToHost);

        hipFree(d_temp_mask);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_final_mask, d_combined_mask, row_count * sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_combined_mask);

    size_t filtered_row_count = 0;
    for (size_t i = 0; i < row_count; i++)
    {
        if (h_final_mask[i])
            filtered_row_count++;
    }

    // std::cout << "Total rows matched: " << filtered_row_count << "\n";
    return h_final_mask;
}

TableResults Filter::applyFilter(const TableResults &input_table) const
{
    if (input_table.row_count == 0)
    {
        return input_table;
    }
    std::unique_ptr<bool[], void (*)(bool *)> h_selected_rows(
        getSelectedRows(input_table),
        [](bool *ptr)
        { delete[] ptr; });
    size_t selected_count = 0;
    const size_t row_count = input_table.row_count;
    for (size_t i = 0; i < row_count; i++)
    {
        if (h_selected_rows[i])
            selected_count++;
    }

    TableResults filtered_table;
    filtered_table.has_more = input_table.has_more;
    filtered_table.column_count = input_table.column_count;
    filtered_table.columns = input_table.columns;
    filtered_table.row_count = selected_count;
    filtered_table.batch_index = input_table.batch_index;
    filtered_table.data.resize(input_table.column_count);

    const size_t chunk_size =10000 ;
    size_t num_chunks = (row_count + chunk_size - 1) / chunk_size;
    std::vector<hipStream_t> streams(std::max<size_t>(1, num_chunks));
    for (auto &stream : streams)
    {
        hipStreamCreate(&stream);
    }
    hipStream_t prefix_stream;
    hipStreamCreate(&prefix_stream);

    bool *d_mask = nullptr;
    hipMalloc(&d_mask, row_count * sizeof(bool));
    hipMemcpyAsync(d_mask, h_selected_rows.get(), row_count * sizeof(bool), hipMemcpyHostToDevice, prefix_stream);

    std::vector<unsigned int> h_positions(row_count);
    unsigned int *d_positions = nullptr;
    hipMalloc(&d_positions, row_count * sizeof(unsigned int));

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; chunk_idx++)
    {
        size_t chunk_offset = chunk_idx * chunk_size;
        size_t chunk_rows = std::min(chunk_size, row_count - chunk_offset);
        int threads = 256;
        int blocks = (chunk_rows + threads - 1) / threads;
        size_t shared_mem = threads * sizeof(unsigned int);
        size_t stream_idx = num_chunks > 1 ? chunk_idx : 0;

        bool *d_chunk_mask = nullptr;
        hipMalloc(&d_chunk_mask, chunk_rows * sizeof(bool));
        hipMemcpyAsync(d_chunk_mask, h_selected_rows.get() + chunk_offset, chunk_rows * sizeof(bool),
                        hipMemcpyHostToDevice, streams[stream_idx]);

        unsigned int *d_chunk_positions = nullptr;
        hipMalloc(&d_chunk_positions, chunk_rows * sizeof(unsigned int));
        computeOutputPositions<<<blocks, threads, shared_mem, streams[stream_idx]>>>(d_chunk_mask, d_chunk_positions, chunk_rows);
        hipMemcpyAsync(h_positions.data() + chunk_offset, d_chunk_positions, chunk_rows * sizeof(unsigned int),
                        hipMemcpyDeviceToHost, streams[stream_idx]);

        hipStreamSynchronize(streams[stream_idx]);
        hipFree(d_chunk_mask);
        hipFree(d_chunk_positions);
    }

    hipMemcpyAsync(d_positions, h_positions.data(), row_count * sizeof(unsigned int), hipMemcpyHostToDevice, prefix_stream);
    hipStreamSynchronize(prefix_stream);

    for (size_t col_idx = 0; col_idx < input_table.column_count; col_idx++)
    {
        const DataType col_type = input_table.columns[col_idx].type;

        switch (col_type)
        {
        case DataType::FLOAT:
        {
            float *h_input_data = static_cast<float *>(input_table.data[col_idx]);
            std::unique_ptr<float[], void (*)(float *)> h_output_data(
                static_cast<float *>(malloc(selected_count * sizeof(float))),
                [](float *ptr)
                { free(ptr); });
            size_t output_offset = 0;

            for (size_t chunk_idx = 0; chunk_idx < num_chunks; chunk_idx++)
            {
                size_t chunk_offset = chunk_idx * chunk_size;
                size_t chunk_rows = std::min(chunk_size, row_count - chunk_offset);
                size_t stream_idx = num_chunks > 1 ? chunk_idx : 0;

                float *d_input = nullptr;
                hipMalloc(&d_input, chunk_rows * sizeof(float));
                hipMemcpyAsync(d_input, h_input_data + chunk_offset, chunk_rows * sizeof(float),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                bool *d_chunk_mask = nullptr;
                hipMalloc(&d_chunk_mask, chunk_rows * sizeof(bool));
                hipMemcpyAsync(d_chunk_mask, h_selected_rows.get() + chunk_offset, chunk_rows * sizeof(bool),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                unsigned int *d_chunk_positions = nullptr;
                hipMalloc(&d_chunk_positions, chunk_rows * sizeof(unsigned int));
                hipMemcpyAsync(d_chunk_positions, h_positions.data() + chunk_offset, chunk_rows * sizeof(unsigned int),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                size_t chunk_selected_count = 0;
                for (size_t i = chunk_offset; i < chunk_offset + chunk_rows && i < row_count; i++)
                {
                    if (h_selected_rows[i])
                        chunk_selected_count++;
                }

                float *d_output = nullptr;
                if (chunk_selected_count > 0)
                {
                    hipMalloc(&d_output, chunk_selected_count * sizeof(float));
                    int threads = 256;
                    int blocks = (chunk_rows + threads - 1) / threads;
                    copySelectedRowsKernel<float><<<blocks, threads, 0, streams[stream_idx]>>>(d_input, d_output, d_chunk_mask, d_chunk_positions, chunk_rows);
                    hipMemcpyAsync(h_output_data.get() + output_offset, d_output, chunk_selected_count * sizeof(float),
                                    hipMemcpyDeviceToHost, streams[stream_idx]);
                    output_offset += chunk_selected_count;
                }

                hipStreamSynchronize(streams[stream_idx]);
                hipFree(d_input);
                if (d_output)
                    hipFree(d_output);
                hipFree(d_chunk_mask);
                hipFree(d_chunk_positions);
            }

            filtered_table.data[col_idx] = h_output_data.release();
            break;
        }
        case DataType::DATETIME:
        {
            uint64_t *h_input_data = static_cast<uint64_t *>(input_table.data[col_idx]);
            std::unique_ptr<uint64_t[], void (*)(uint64_t *)> h_output_data(
                static_cast<uint64_t *>(malloc(selected_count * sizeof(uint64_t))),
                [](uint64_t *ptr)
                { free(ptr); });
            size_t output_offset = 0;

            for (size_t chunk_idx = 0; chunk_idx < num_chunks; chunk_idx++)
            {
                size_t chunk_offset = chunk_idx * chunk_size;
                size_t chunk_rows = std::min(chunk_size, row_count - chunk_offset);
                size_t stream_idx = num_chunks > 1 ? chunk_idx : 0;

                uint64_t *d_input = nullptr;
                hipMalloc(&d_input, chunk_rows * sizeof(uint64_t));
                hipMemcpyAsync(d_input, h_input_data + chunk_offset, chunk_rows * sizeof(uint64_t),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                bool *d_chunk_mask = nullptr;
                hipMalloc(&d_chunk_mask, chunk_rows * sizeof(bool));
                hipMemcpyAsync(d_chunk_mask, h_selected_rows.get() + chunk_offset, chunk_rows * sizeof(bool),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                unsigned int *d_chunk_positions = nullptr;
                hipMalloc(&d_chunk_positions, chunk_rows * sizeof(unsigned int));
                hipMemcpyAsync(d_chunk_positions, h_positions.data() + chunk_offset, chunk_rows * sizeof(unsigned int),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                size_t chunk_selected_count = 0;
                for (size_t i = chunk_offset; i < chunk_offset + chunk_rows && i < row_count; i++)
                {
                    if (h_selected_rows[i])
                        chunk_selected_count++;
                }

                uint64_t *d_output = nullptr;
                if (chunk_selected_count > 0)
                {
                    hipMalloc(&d_output, chunk_selected_count * sizeof(uint64_t));
                    int threads = 256;
                    int blocks = (chunk_rows + threads - 1) / threads;
                    copySelectedRowsKernel<uint64_t><<<blocks, threads, 0, streams[stream_idx]>>>(d_input, d_output, d_chunk_mask, d_chunk_positions, chunk_rows);
                    hipMemcpyAsync(h_output_data.get() + output_offset, d_output, chunk_selected_count * sizeof(uint64_t),
                                    hipMemcpyDeviceToHost, streams[stream_idx]);
                    output_offset += chunk_selected_count;
                }

                hipStreamSynchronize(streams[stream_idx]);
                hipFree(d_input);
                if (d_output)
                    hipFree(d_output);
                hipFree(d_chunk_mask);
                hipFree(d_chunk_positions);
            }

            filtered_table.data[col_idx] = h_output_data.release();
            break;
        }
        case DataType::STRING:
        {
            const char **h_input_strings = static_cast<const char **>(input_table.data[col_idx]);
            std::unique_ptr<const char *[], void (*)(const char **)> h_output_strings(
                static_cast<const char **>(malloc(selected_count * sizeof(char *))),
                [](const char **ptr)
                { free(ptr); });
            size_t output_offset = 0;

            for (size_t chunk_idx = 0; chunk_idx < num_chunks; chunk_idx++)
            {
                size_t chunk_offset = chunk_idx * chunk_size;
                size_t chunk_rows = std::min(chunk_size, row_count - chunk_offset);
                size_t stream_idx = num_chunks > 1 ? chunk_idx : 0;

                const char **d_input_strings = nullptr;
                hipMalloc(&d_input_strings, chunk_rows * sizeof(char *));
                hipMemcpyAsync(d_input_strings, h_input_strings + chunk_offset, chunk_rows * sizeof(char *),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                bool *d_chunk_mask = nullptr;
                hipMalloc(&d_chunk_mask, chunk_rows * sizeof(bool));
                hipMemcpyAsync(d_chunk_mask, h_selected_rows.get() + chunk_offset, chunk_rows * sizeof(bool),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                unsigned int *d_chunk_positions = nullptr;
                hipMalloc(&d_chunk_positions, chunk_rows * sizeof(unsigned int));
                hipMemcpyAsync(d_chunk_positions, h_positions.data() + chunk_offset, chunk_rows * sizeof(unsigned int),
                                hipMemcpyHostToDevice, streams[stream_idx]);

                size_t chunk_selected_count = 0;
                for (size_t i = chunk_offset; i < chunk_offset + chunk_rows && i < row_count; i++)
                {
                    if (h_selected_rows[i])
                        chunk_selected_count++;
                }

                const char **d_output_strings = nullptr;
                if (chunk_selected_count > 0)
                {
                    hipMalloc(&d_output_strings, chunk_selected_count * sizeof(char *));
                    int threads = 256;
                    int blocks = (chunk_rows + threads - 1) / threads;
                    copySelectedStringRowsKernel<<<blocks, threads, 0, streams[stream_idx]>>>(d_input_strings, d_output_strings, d_chunk_mask, d_chunk_positions, chunk_rows);
                    hipMemcpyAsync(h_output_strings.get() + output_offset, d_output_strings, chunk_selected_count * sizeof(char *),
                                    hipMemcpyDeviceToHost, streams[stream_idx]);
                    output_offset += chunk_selected_count;
                }

                hipStreamSynchronize(streams[stream_idx]);
                hipFree(d_input_strings);
                if (d_output_strings)
                    hipFree(d_output_strings);
                hipFree(d_chunk_mask);
                hipFree(d_chunk_positions);
            }

            filtered_table.data[col_idx] = h_output_strings.release();
            break;
        }
        default:
            for (auto &stream : streams)
                hipStreamDestroy(stream);
            hipStreamDestroy(prefix_stream);
            hipFree(d_mask);
            hipFree(d_positions);
            throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(col_type)));
        }
    }

    for (auto &stream : streams)
    {
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);
    }
    hipStreamDestroy(prefix_stream);
    hipFree(d_mask);
    hipFree(d_positions);

    return filtered_table;
}
void Filter::print() const
{
    std::cout << "FILTER (";
    for (size_t i = 0; i < conditions.size(); ++i)
    {
        if (i > 0 && i - 1 < logical_ops.size())
        {
            std::cout << " " << logical_ops[i - 1] << " ";
        }
        std::cout << conditions[i].column << " " << conditions[i].op << " ";
        if (conditions[i].is_string)
        {
            std::cout << "'" << conditions[i].value << "'";
        }
        else
        {
            std::cout << conditions[i].value;
        }
    }
    std::cout << ")\n";
}
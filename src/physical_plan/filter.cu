#include "physical_plan/filter.hpp"

Filter::Filter(const duckdb::InsertionOrderPreservingMap<std::string> &params)
    : PhysicalOpNode()
{
    auto it = params.find("__expression__");
    if (it != params.end())
    {
        parseConditions(it->second);
        removeTimestampSuffixSimple();
    }
}
void Filter::removeTimestampSuffixSimple()
{
    const std::string suffix = "::TIMESTAMP";

    for (auto &cond : conditions)
    {
        size_t pos = cond.value.rfind(suffix);
        if (pos != std::string::npos)
        {
            cond.value = cond.value.substr(1, pos - 1);
        }
    }
}
std::string Filter::trim(const std::string &str) const
{
    size_t first = str.find_first_not_of(" \t\n\r");
    if (first == std::string::npos)
        return "";
    size_t last = str.find_last_not_of(" \t\n\r");
    return str.substr(first, (last - first + 1));
}

void Filter::parseConditions(const std::string &expression)
{
    std::string expr = trim(expression);

    size_t and_pos = expr.find(" AND ");
    size_t or_pos = expr.find(" OR ");

    if (and_pos != std::string::npos || or_pos != std::string::npos)
    {
        parseComplexCondition(expr);
    }
    else
    {
        parseSimpleCondition(expr);
    }
}

void Filter::parseSimpleCondition(const std::string &expr)
{
    std::string inner = trim(expr);
    if (inner.front() == '(' && inner.back() == ')')
    {
        inner = inner.substr(1, inner.length() - 2);
    }

    Condition cond = parseSingleCondition(inner);
    if (!cond.column.empty())
    {
        conditions.push_back(cond);
    }
}

void Filter::parseComplexCondition(const std::string &expr)
{
    std::string inner = trim(expr);
    if (inner.front() == '(' && inner.back() == ')')
    {
        inner = inner.substr(1, inner.length() - 2);
    }

    std::vector<std::string> tokens;
    std::string current;
    int paren_level = 0;

    for (char c : inner)
    {
        if (c == '(')
            paren_level++;
        if (c == ')')
            paren_level--;

        if (paren_level == 0 && c == ' ')
        {
            if (!current.empty())
            {
                tokens.push_back(current);
                current.clear();
            }
        }
        else
        {
            current += c;
        }
    }
    if (!current.empty())
        tokens.push_back(current);

    for (size_t i = 0; i < tokens.size();)
    {
        if (tokens[i] == "AND" || tokens[i] == "OR")
        {
            logical_ops.push_back(tokens[i]);
            i++;
        }
        else
        {
            parseSimpleCondition(tokens[i]);
            i++;
        }
    }
}

Condition Filter::parseSingleCondition(const std::string &cond_expr) const
{
    Condition cond;
    std::regex op_regex(R"((>=|<=|!=|>|<|=))");
    std::smatch op_match;

    if (std::regex_search(cond_expr, op_match, op_regex))
    {
        cond.column = trim(cond_expr.substr(0, op_match.position()));
        cond.op = op_match.str();
        cond.value = trim(cond_expr.substr(op_match.position() + cond.op.length()));

        if (!cond.value.empty() && cond.value.front() == '\'' && cond.value.back() == '\'')
        {
            cond.value = cond.value.substr(1, cond.value.length() - 2);
            cond.is_string = true;
        }
        else
        {
            cond.is_string = false;
        }
    }
    return cond;
}

bool *Filter::getSelectedRows(const TableResults &input_table) const
{
    const size_t row_count = input_table.row_count;
    bool *h_final_mask = new bool[row_count]();

    bool *d_combined_mask = nullptr;
    hipMalloc(&d_combined_mask, row_count * sizeof(bool));
    hipMemset(d_combined_mask, 0, row_count * sizeof(bool));

    int numThreads = 256;
    int numBlocks = (row_count + numThreads - 1) / numThreads;

    for (size_t cond_idx = 0; cond_idx < this->conditions.size(); cond_idx++)
    {
        const auto &cond = this->conditions[cond_idx];
        const size_t col_idx = input_table.getColumnIndex(cond.column);
        const DataType col_type = input_table.columns[col_idx].type;

        uint8_t cond_code = 0;
        if (cond.op == ">")
            cond_code = 1;
        else if (cond.op == "<")
            cond_code = 2;
        else if (cond.op == "=")
            cond_code = 3;
        else if (cond.op == "!=")
            cond_code = 4;
        else if (cond.op == "<=")
            cond_code = 5;
        else if (cond.op == ">=")
            cond_code = 6;
        else
        {
            std::cerr << "Unsupported operator: " << cond.op << "\n";
            continue;
        }

        bool *d_temp_mask = nullptr;
        hipMalloc(&d_temp_mask, row_count * sizeof(bool));
        hipMemset(d_temp_mask, 0, row_count * sizeof(bool));

        if (col_type == DataType::FLOAT)
        {
            float *d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(float));
            hipMemcpy(d_col_data, input_table.data[col_idx], row_count * sizeof(float), hipMemcpyHostToDevice);
            float value = std::stof(cond.value);
            filterKernel<float><<<numBlocks, numThreads>>>(d_col_data, d_temp_mask, row_count, value, cond_code);
            hipFree(d_col_data);
        }
        else if (col_type == DataType::DATETIME)
        {
            uint64_t *d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(uint64_t));
            hipMemcpy(d_col_data, input_table.data[col_idx], row_count * sizeof(uint64_t), hipMemcpyHostToDevice);
            uint64_t value = getDateTime(cond.value);
            filterKernel<uint64_t><<<numBlocks, numThreads>>>(d_col_data, d_temp_mask, row_count, value, cond_code);
            hipFree(d_col_data);
        }
        else if (col_type == DataType::STRING)
        {
            const char **d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(char *));
            const char **host_strings = static_cast<const char **>(input_table.data[col_idx]);

            char **d_strings = new char *[row_count];

            for (size_t i = 0; i < row_count; i++)
            {
                size_t len = strlen(host_strings[i]) + 1;
                hipMalloc(&d_strings[i], len);
                hipMemcpy(d_strings[i], host_strings[i], len, hipMemcpyHostToDevice);
                hipMemcpy(&d_col_data[i], &d_strings[i], sizeof(char *), hipMemcpyHostToDevice);
            }

            char *d_value = nullptr;
            hipMalloc(&d_value, cond.value.size() + 1);
            hipMemcpy(d_value, cond.value.c_str(), cond.value.size() + 1, hipMemcpyHostToDevice);
            filterKernelString<<<numBlocks, numThreads>>>(d_col_data, d_temp_mask, row_count, d_value, cond_code);

            for (size_t i = 0; i < row_count; i++)
            {
                hipFree(d_strings[i]);
            }
            delete[] d_strings;
            hipFree(d_value);
            hipFree(d_col_data);
        }

        if (cond_idx == 0)
        {
            hipMemcpy(d_combined_mask, d_temp_mask, row_count * sizeof(bool), hipMemcpyDeviceToDevice);
        }
        else
        {
            const std::string &op = logical_ops[cond_idx - 1];
            if (op == "AND")
            {
                andKernel<<<numBlocks, numThreads>>>(d_combined_mask, d_temp_mask, row_count);
            }
            else if (op == "OR")
            {
                orKernel<<<numBlocks, numThreads>>>(d_combined_mask, d_temp_mask, row_count);
            }
        }
        bool *h_temp_mask = new bool[row_count];
        hipMemcpy(h_temp_mask, d_temp_mask, row_count * sizeof(bool), hipMemcpyDeviceToHost);

        hipFree(d_temp_mask);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_final_mask, d_combined_mask, row_count * sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_combined_mask);

    size_t filtered_row_count = 0;
    for (size_t i = 0; i < row_count; i++)
    {
        if (h_final_mask[i])
            filtered_row_count++;
    }

    // std::cout << "Total rows matched: " << filtered_row_count << "\n";
    return h_final_mask;
}

TableResults Filter::applyFilter(const TableResults &input_table) const
{
    if (input_table.row_count == 0)
    {
        return input_table;
    }
    std::unique_ptr<bool[], void (*)(bool *)> h_selected_rows(
        getSelectedRows(input_table),
        [](bool *ptr)
        { delete[] ptr; });
    size_t selected_count = 0;
    const size_t row_count = input_table.row_count;
    for (size_t i = 0; i < row_count; i++)
    {
        if (h_selected_rows[i])
            selected_count++;
    }

    TableResults filtered_table;
    filtered_table.has_more = input_table.has_more;
    filtered_table.column_count = input_table.column_count;
    filtered_table.columns = input_table.columns;
    filtered_table.row_count = selected_count;
    filtered_table.batch_index = input_table.batch_index;
    filtered_table.data.resize(input_table.column_count);

    // Create streams: one for prefix sum and one per column
    hipStream_t prefix_stream;
    hipStreamCreate(&prefix_stream);
    std::vector<hipStream_t> col_streams(input_table.column_count);
    for (auto &stream : col_streams)
    {
        hipStreamCreate(&stream);
    }

    bool *d_mask = nullptr;
    unsigned int *d_positions;

    hipMalloc(&d_mask, row_count * sizeof(bool));
    hipMalloc(&d_positions, row_count * sizeof(unsigned int));

    hipMemcpyAsync(d_mask, h_selected_rows.get(), row_count * sizeof(bool), hipMemcpyHostToDevice, prefix_stream);

    int threads = 256;
    int blocks = (row_count + threads - 1) / threads;
    size_t shared_mem = threads * sizeof(unsigned int);

    computeOutputPositions<<<blocks, threads, shared_mem, prefix_stream>>>(d_mask, d_positions, row_count);
    hipStreamSynchronize(prefix_stream);

    for (size_t col_idx = 0; col_idx < input_table.column_count; col_idx++)
    {
        const DataType col_type = input_table.columns[col_idx].type;

        switch (col_type)
        {
        case DataType::FLOAT:
        {
            float *h_input_data = static_cast<float *>(input_table.data[col_idx]);
            float *d_input = nullptr;
            float *d_output = nullptr;
            hipMalloc(&d_input, row_count * sizeof(float));
            hipMalloc(&d_output, selected_count * sizeof(float));
            std::unique_ptr<float[], void (*)(float *)> h_output_data(
                static_cast<float *>(malloc(selected_count * sizeof(float))),
                [](float *ptr)
                { free(ptr); });

            hipMemcpyAsync(d_input, h_input_data, row_count * sizeof(float), hipMemcpyHostToDevice, col_streams[col_idx]);
            copySelectedRowsKernel<float><<<blocks, threads, 0, col_streams[col_idx]>>>(d_input, d_output, d_mask, d_positions, row_count);
            hipMemcpyAsync(h_output_data.get(), d_output, selected_count * sizeof(float), hipMemcpyDeviceToHost, col_streams[col_idx]);

            filtered_table.data[col_idx] = h_output_data.release();
            hipFree(d_input);
            hipFree(d_output);
            break;
        }
        case DataType::DATETIME:
        {
            uint64_t *h_input_data = static_cast<uint64_t *>(input_table.data[col_idx]);
            uint64_t *d_input = nullptr;
            uint64_t *d_output = nullptr;
            hipMalloc(&d_input, row_count * sizeof(uint64_t));
            hipMalloc(&d_output, selected_count * sizeof(uint64_t));
            std::unique_ptr<uint64_t[], void (*)(uint64_t *)> h_output_data(
                static_cast<uint64_t *>(malloc(selected_count * sizeof(uint64_t))),
                [](uint64_t *ptr)
                { free(ptr); });

            hipMemcpyAsync(d_input, h_input_data, row_count * sizeof(uint64_t), hipMemcpyHostToDevice, col_streams[col_idx]);
            copySelectedRowsKernel<uint64_t><<<blocks, threads, 0, col_streams[col_idx]>>>(d_input, d_output, d_mask, d_positions, row_count);
            hipMemcpyAsync(h_output_data.get(), d_output, selected_count * sizeof(uint64_t), hipMemcpyDeviceToHost, col_streams[col_idx]);

            filtered_table.data[col_idx] = h_output_data.release();
            hipFree(d_input);
            hipFree(d_output);
            break;
        }
        case DataType::STRING:
        {
            const char **h_input_strings = static_cast<const char **>(input_table.data[col_idx]);
            std::unique_ptr<const char *[], void (*)(const char **)>
                h_output_strings(
                    static_cast<const char **>(malloc(selected_count * sizeof(char *))),
                    [](const char **ptr)
                    { free(ptr); });

            const char **d_input_strings = nullptr;
            hipMalloc(&d_input_strings, row_count * sizeof(char *));
            hipMemcpyAsync(d_input_strings, h_input_strings, row_count * sizeof(char *), hipMemcpyHostToDevice, col_streams[col_idx]);

            const char **d_output_strings = nullptr;
            hipMalloc(&d_output_strings, selected_count * sizeof(char *));

            copySelectedStringRowsKernel<<<blocks, threads, 0, col_streams[col_idx]>>>(d_input_strings, d_output_strings, d_mask, d_positions, row_count);
            hipMemcpyAsync(h_output_strings.get(), d_output_strings, selected_count * sizeof(char *), hipMemcpyDeviceToHost, col_streams[col_idx]);

            filtered_table.data[col_idx] = h_output_strings.release();
            hipFree(d_input_strings);
            hipFree(d_output_strings);
            break;
        }
        default:
            hipStreamDestroy(prefix_stream);
            for (auto &stream : col_streams)
                hipStreamDestroy(stream);
            hipFree(d_mask);
            hipFree(d_positions);
            throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(col_type)));
        }
    }

    for (auto &stream : col_streams)
    {
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);
    }
    hipStreamDestroy(prefix_stream);
    hipFree(d_mask);
    hipFree(d_positions);

    return filtered_table;
}
void Filter::print() const
{
    std::cout << "FILTER (";
    for (size_t i = 0; i < conditions.size(); ++i)
    {
        if (i > 0 && i - 1 < logical_ops.size())
        {
            std::cout << " " << logical_ops[i - 1] << " ";
        }
        std::cout << conditions[i].column << " " << conditions[i].op << " ";
        if (conditions[i].is_string)
        {
            std::cout << "'" << conditions[i].value << "'";
        }
        else
        {
            std::cout << conditions[i].value;
        }
    }
    std::cout << ")\n";
}
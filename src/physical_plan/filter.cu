#include "physical_plan/filter.hpp"

Filter::Filter(const duckdb::InsertionOrderPreservingMap<std::string> &params)
    : PhysicalOpNode()
{
    auto it = params.find("__expression__");
    if (it != params.end())
    {
        parseConditions(it->second);
        removeTimestampSuffixSimple();
    }
}
void Filter::removeTimestampSuffixSimple()
{
    const std::string suffix = "::TIMESTAMP";

    for (auto &cond : conditions)
    {
        size_t pos = cond.value.rfind(suffix);
        if (pos != std::string::npos)
        {
            cond.value = cond.value.substr(1, pos - 1);
        }
    }
}
std::string Filter::trim(const std::string &str) const
{
    size_t first = str.find_first_not_of(" \t\n\r");
    if (first == std::string::npos)
        return "";
    size_t last = str.find_last_not_of(" \t\n\r");
    return str.substr(first, (last - first + 1));
}

void Filter::parseConditions(const std::string &expression)
{
    std::string expr = trim(expression);

    size_t and_pos = expr.find(" AND ");
    size_t or_pos = expr.find(" OR ");

    if (and_pos != std::string::npos || or_pos != std::string::npos)
    {
        parseComplexCondition(expr);
    }
    else
    {
        parseSimpleCondition(expr);
    }
}

void Filter::parseSimpleCondition(const std::string &expr)
{
    std::string inner = trim(expr);
    if (inner.front() == '(' && inner.back() == ')')
    {
        inner = inner.substr(1, inner.length() - 2);
    }

    Condition cond = parseSingleCondition(inner);
    if (!cond.column.empty())
    {
        conditions.push_back(cond);
    }
}

void Filter::parseComplexCondition(const std::string &expr)
{
    std::string inner = trim(expr);
    if (inner.front() == '(' && inner.back() == ')')
    {
        inner = inner.substr(1, inner.length() - 2);
    }

    std::vector<std::string> tokens;
    std::string current;
    int paren_level = 0;

    for (char c : inner)
    {
        if (c == '(')
            paren_level++;
        if (c == ')')
            paren_level--;

        if (paren_level == 0 && c == ' ')
        {
            if (!current.empty())
            {
                tokens.push_back(current);
                current.clear();
            }
        }
        else
        {
            current += c;
        }
    }
    if (!current.empty())
        tokens.push_back(current);

    for (size_t i = 0; i < tokens.size();)
    {
        if (tokens[i] == "AND" || tokens[i] == "OR")
        {
            logical_ops.push_back(tokens[i]);
            i++;
        }
        else
        {
            parseSimpleCondition(tokens[i]);
            i++;
        }
    }
}

Condition Filter::parseSingleCondition(const std::string &cond_expr) const
{
    Condition cond;
    std::regex op_regex(R"((>=|<=|!=|>|<|=))");
    std::smatch op_match;

    if (std::regex_search(cond_expr, op_match, op_regex))
    {
        cond.column = trim(cond_expr.substr(0, op_match.position()));
        cond.op = op_match.str();
        cond.value = trim(cond_expr.substr(op_match.position() + cond.op.length()));

        if (!cond.value.empty() && cond.value.front() == '\'' && cond.value.back() == '\'')
        {
            cond.value = cond.value.substr(1, cond.value.length() - 2);
            cond.is_string = true;
        }
        else
        {
            cond.is_string = false;
        }
    }
    return cond;
}

enum class ConditionOp
{
    Greater = 1,
    Less = 2,
    Equal = 3,
    NotEqual = 4,
    LessEqual = 5,
    GreaterEqual = 6
};

ConditionOp getConditionCode(const std::string &op)
{
    if (op == ">")
        return ConditionOp::Greater;
    if (op == "<")
        return ConditionOp::Less;
    if (op == "=")
        return ConditionOp::Equal;
    if (op == "!=")
        return ConditionOp::NotEqual;
    if (op == "<=")
        return ConditionOp::LessEqual;
    if (op == ">=")
        return ConditionOp::GreaterEqual;
    throw std::invalid_argument("Unsupported operator: " + op);
}

bool *Filter::getSelectedRows(const TableResults &input_table) const
{
    const size_t row_count = input_table.row_count;
    std::unique_ptr<bool[], void (*)(bool *)> h_final_mask(
        new bool[row_count](),
        [](bool *ptr)
        { delete[] ptr; });
    std::vector<hipStream_t> streams(conditions.size());
    for (auto &stream : streams)
    {
        hipStreamCreate(&stream);
    }
    hipStream_t final_stream;
    hipStreamCreate(&final_stream);

    bool *d_combined_mask = nullptr;
    hipMalloc(&d_combined_mask, row_count * sizeof(bool));
    hipMemsetAsync(d_combined_mask, 0, row_count * sizeof(bool), final_stream);

    int numThreads = 256;
    int numBlocks = (row_count + numThreads - 1) / numThreads;
    std::vector<bool *> d_temp_masks(conditions.size());

    for (size_t cond_idx = 0; cond_idx < this->conditions.size(); cond_idx++)
    {

        bool *d_temp_mask = nullptr;
        hipMalloc(&d_temp_masks[cond_idx], row_count * sizeof(bool));
        hipMemsetAsync(&d_temp_masks[cond_idx], 0, row_count * sizeof(bool), streams[cond_idx]);

        const auto &cond = this->conditions[cond_idx];
        const size_t col_idx = input_table.getColumnIndex(cond.column);
        const DataType col_type = input_table.columns[col_idx].type;
        ConditionOp cond_code = getConditionCode(cond.op);

        switch (col_type)
        {
        case DataType::FLOAT:
        {
            float *d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(float));
            hipMemcpyAsync(d_col_data, input_table.data[col_idx], row_count * sizeof(float), hipMemcpyHostToDevice, streams[cond_idx]);
            float value = std::stof(cond.value);
            filterKernel<float><<<numBlocks, numThreads, 0, streams[cond_idx]>>>(d_col_data, d_temp_masks[cond_idx], row_count, value, static_cast<uint8_t>(cond_code));
            hipFree(d_col_data);
            break;
        }
        case DataType::DATETIME:
        {
            uint64_t *d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(uint64_t));
            hipMemcpyAsync(d_col_data, input_table.data[col_idx], row_count * sizeof(uint64_t), hipMemcpyHostToDevice, streams[cond_idx]);
            uint64_t value = getDateTime(cond.value);
            filterKernel<uint64_t><<<numBlocks, numThreads, 0, streams[cond_idx]>>>(d_col_data, d_temp_masks[cond_idx], row_count, value, static_cast<uint8_t>(cond_code));
            hipFree(d_col_data);
            break;
        }
        case DataType::STRING:
        {
            const char **d_col_data = nullptr;
            hipMalloc(&d_col_data, row_count * sizeof(char *));
            const char **host_strings = static_cast<const char **>(input_table.data[col_idx]);

            std::unique_ptr<char *[], void (*)(char **)>
            d_strings(
                new char *[row_count],
                [](char **ptr)
                { delete[] ptr; });

            for (size_t i = 0; i < row_count; i++)
            {
                size_t len = strlen(host_strings[i]) + 1;
                hipMalloc(&d_strings[i], len);
                hipMemcpyAsync(d_strings[i], host_strings[i], len, hipMemcpyHostToDevice, streams[cond_idx]);
                hipMemcpyAsync(&d_col_data[i], &d_strings[i], sizeof(char *), hipMemcpyHostToDevice, streams[cond_idx]);
            }

            char *d_value = nullptr;
            hipMalloc(&d_value, cond.value.size() + 1);
            hipMemcpyAsync(d_value, cond.value.c_str(), cond.value.size() + 1, hipMemcpyHostToDevice, streams[cond_idx]);
            filterKernelString<<<numBlocks, numThreads, 0, streams[cond_idx]>>>(d_col_data, d_temp_masks[cond_idx], row_count, d_value, static_cast<uint8_t>(cond_code));

            for (size_t i = 0; i < row_count; i++)
            {
                hipFree(d_strings[i]);
            }
            hipFree(d_value);
            hipFree(d_col_data);
            break;
        }
        default:
            for (auto &stream : streams)
                hipStreamDestroy(stream);
            hipStreamDestroy(final_stream);
            hipFree(d_combined_mask);
            for (auto &mask : d_temp_masks)
                hipFree(mask);
            throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(col_type)));
        }
        hipStreamSynchronize(streams[cond_idx]);

        if (cond_idx == 0)
        {
            hipMemcpyAsync(d_combined_mask, d_temp_masks[cond_idx], row_count * sizeof(bool), hipMemcpyDeviceToDevice, final_stream);
        }
        else
        {
            const std::string &op = logical_ops[cond_idx - 1];
            if (op == "AND")
            {
                andKernel<<<numBlocks, numThreads, 0, final_stream>>>(d_combined_mask, d_temp_mask, row_count);
            }
            else if (op == "OR")
            {
                orKernel<<<numBlocks, numThreads, 0, final_stream>>>(d_combined_mask, d_temp_mask, row_count);
            }
        }
        hipFree(d_temp_masks[cond_idx]);
    }

    hipMemcpyAsync(h_final_mask.get(), d_combined_mask, row_count * sizeof(bool), hipMemcpyDeviceToHost, final_stream);
    hipStreamSynchronize(final_stream);
    hipFree(d_combined_mask);

    for (auto &stream : streams)
    {
        hipStreamDestroy(stream);
    }
    hipStreamDestroy(final_stream);

    size_t filtered_row_count = 0;
    for (size_t i = 0; i < row_count; i++)
    {
        if (h_final_mask[i])
            filtered_row_count++;
    }

    return h_final_mask.release();
}

TableResults Filter::applyFilter(const TableResults &input_table) const
{
    if (input_table.row_count == 0)
    {
        return input_table;
    }
    bool *h_selected_rows = getSelectedRows(input_table);

    size_t selected_count = 0;
    const size_t row_count = input_table.row_count;
    for (size_t i = 0; i < row_count; i++)
    {
        if (h_selected_rows[i])
            selected_count++;
    }

    TableResults filtered_table;
    filtered_table.has_more = input_table.has_more;
    filtered_table.column_count = input_table.column_count;
    filtered_table.columns = input_table.columns;
    filtered_table.row_count = selected_count;
    filtered_table.batch_index = input_table.batch_index;
    filtered_table.data.resize(input_table.column_count);

    bool *d_mask;
    hipMalloc(&d_mask, row_count * sizeof(bool));
    hipMemcpy(d_mask, h_selected_rows, row_count * sizeof(bool), hipMemcpyHostToDevice);

    unsigned int *d_positions;
    hipMalloc(&d_positions, row_count * sizeof(unsigned int));

    int threads = 256;
    int blocks = (row_count + threads - 1) / threads;
    size_t shared_mem = threads * sizeof(unsigned int);

    computeOutputPositions<<<blocks, threads, shared_mem>>>(d_mask, d_positions, row_count);
    hipDeviceSynchronize();

    for (size_t col_idx = 0; col_idx < input_table.column_count; col_idx++)
    {
        const DataType col_type = input_table.columns[col_idx].type;

        if (col_type == DataType::FLOAT)
        {
            float *h_input_data = static_cast<float *>(input_table.data[col_idx]);
            float *d_input, *d_output;
            hipMalloc(&d_input, row_count * sizeof(float));
            hipMalloc(&d_output, selected_count * sizeof(float));

            float *h_output_data = static_cast<float *>(malloc(selected_count * sizeof(float)));
            hipMemcpy(d_input, h_input_data, row_count * sizeof(float), hipMemcpyHostToDevice);
            copySelectedRowsKernel<float><<<blocks, threads>>>(d_input, d_output, d_mask, d_positions, row_count);
            hipDeviceSynchronize();
            hipMemcpy(h_output_data, d_output, selected_count * sizeof(float), hipMemcpyDeviceToHost);

            filtered_table.data[col_idx] = h_output_data;
            hipFree(d_input);
            hipFree(d_output);
        }
        else if (col_type == DataType::DATETIME)
        {
            uint64_t *h_input_data = static_cast<uint64_t *>(input_table.data[col_idx]);
            uint64_t *d_input, *d_output;
            hipMalloc(&d_input, row_count * sizeof(uint64_t));
            hipMalloc(&d_output, selected_count * sizeof(uint64_t));
            uint64_t *h_output_data = static_cast<uint64_t *>(malloc(selected_count * sizeof(uint64_t)));

            hipMemcpy(d_input, h_input_data, row_count * sizeof(uint64_t), hipMemcpyHostToDevice);
            copySelectedRowsKernel<uint64_t><<<blocks, threads>>>(d_input, d_output, d_mask, d_positions, row_count);
            hipDeviceSynchronize();
            hipMemcpy(h_output_data, d_output, selected_count * sizeof(uint64_t), hipMemcpyDeviceToHost);

            filtered_table.data[col_idx] = h_output_data;
            hipFree(d_input);
            hipFree(d_output);
        }
        else if (col_type == DataType::STRING)
        {
            const char **h_input_strings = static_cast<const char **>(input_table.data[col_idx]);
            const char **h_output_strings = static_cast<const char **>(malloc(selected_count * sizeof(char *)));

            const char **d_input_strings;
            hipMalloc(&d_input_strings, row_count * sizeof(char *));
            hipMemcpy(d_input_strings, h_input_strings, row_count * sizeof(char *), hipMemcpyHostToDevice);

            const char **d_output_strings;
            hipMalloc(&d_output_strings, selected_count * sizeof(char *));

            copySelectedStringRowsKernel<<<blocks, threads>>>(d_input_strings, d_output_strings, d_mask, d_positions, row_count);
            hipDeviceSynchronize();

            hipMemcpy(h_output_strings, d_output_strings, selected_count * sizeof(char *), hipMemcpyDeviceToHost);

            filtered_table.data[col_idx] = h_output_strings;
            hipFree(d_input_strings);
            hipFree(d_output_strings);
        }
    }

    hipFree(d_mask);
    hipFree(d_positions);
    delete[] h_selected_rows;

    return filtered_table;
}
void Filter::print() const
{
    std::cout << "FILTER (";
    for (size_t i = 0; i < conditions.size(); ++i)
    {
        if (i > 0 && i - 1 < logical_ops.size())
        {
            std::cout << " " << logical_ops[i - 1] << " ";
        }
        std::cout << conditions[i].column << " " << conditions[i].op << " ";
        if (conditions[i].is_string)
        {
            std::cout << "'" << conditions[i].value << "'";
        }
        else
        {
            std::cout << conditions[i].value;
        }
    }
    std::cout << ")\n";
}
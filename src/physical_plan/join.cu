#include "hip/hip_runtime.h"
#include "physical_plan/join.hpp"

HashJoin::HashJoin(const duckdb::InsertionOrderPreservingMap<std::string> &params) : PhysicalOpNode()
{
    auto it = params.find("Conditions");
    if (it != params.end())
    {
        size_t pos = it->second.find('=');
        if (pos != std::string::npos)
        {
            col_table_left = it->second.substr(0, pos - 1);
            col_table_right = it->second.substr(pos + 2);
        }
        // std::cout << "col_table_left: " << col_table_left << std::endl;
        // std::cout << "col_table_right: " << col_table_right << std::endl;
    }
}

void HashJoin::getIndexOfSelectedRows(const TableResults &left_table, const TableResults &right_table,
                                      std::vector<size_t> &left_indices, std::vector<size_t> &right_indices)
{
    size_t index_left = left_table.getColumnIndex(col_table_left);
    size_t index_right = right_table.getColumnIndex(col_table_right);
    ColumnInfo left_col = left_table.columns[index_left];
    ColumnInfo right_col = right_table.columns[index_right];

    if (left_col.type != right_col.type)
    {
        throw std::runtime_error("Join columns have different types");
    }

    const size_t chunk_size = 10000;
    size_t row_count_left = left_table.row_count;
    size_t row_count_right = right_table.row_count;
    size_t num_chunks = (row_count_right + chunk_size - 1) / chunk_size;
    const size_t max_streams = 16;
    size_t num_streams = std::min(num_chunks, max_streams);
    std::vector<hipStream_t> streams(num_streams);
    std::vector<void *> allocations;

    for (auto &stream : streams)
    {
        hipError_t err = hipStreamCreate(&stream);
    }

    int numThreads = 256;
    size_t shared_mem_size = ((numThreads + 31) / 32) * getDataTypeNumBytes(left_col.type);

    try
    {
        if (left_col.type == DataType::FLOAT)
        {
            float *d_left_data = nullptr;
            hipError_t err = hipMalloc(&d_left_data, row_count_left * sizeof(float));

            allocations.push_back(d_left_data);

            err = hipMemcpy(d_left_data, left_table.data[index_left], row_count_left * sizeof(float), hipMemcpyHostToDevice);

            for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx)
            {
                size_t right_start = chunk_idx * chunk_size;
                size_t right_rows = std::min(chunk_size, row_count_right - right_start);
                if (right_rows == 0)
                    continue;

                hipStream_t stream = streams[chunk_idx % num_streams];

                // Allocate for this batch: max pairs = row_count_left * right_rows
                size_t max_batch_pairs = row_count_left * right_rows;
                size_t *d_left_idx = nullptr;
                size_t *d_right_idx = nullptr;
                unsigned long long *d_count = nullptr;

                err = hipMalloc(&d_left_idx, max_batch_pairs * sizeof(size_t));

                allocations.push_back(d_left_idx);

                err = hipMalloc(&d_right_idx, max_batch_pairs * sizeof(size_t));

                allocations.push_back(d_right_idx);

                err = hipMalloc(&d_count, sizeof(unsigned long long));

                allocations.push_back(d_count);

                err = hipMemset(d_count, 0, sizeof(unsigned long long));

                float *d_right_data = nullptr;
                err = hipMalloc(&d_right_data, right_rows * sizeof(float));

                allocations.push_back(d_right_data);

                err = hipMemcpyAsync(d_right_data,
                                      static_cast<float *>(right_table.data[index_right]) + right_start,
                                      right_rows * sizeof(float), hipMemcpyHostToDevice, stream);

                size_t chunk_blocks = (std::max(row_count_left, right_rows) + numThreads - 1) / numThreads;
                size_t shared_mem_size2 = numThreads * sizeof(float);
                hashJoinKernel<float><<<chunk_blocks, numThreads, shared_mem_size2, stream>>>(
                    d_left_data, d_right_data,
                    row_count_left, right_rows,
                    d_left_idx, d_right_idx, d_count);
                err = hipGetLastError();

                err = hipStreamSynchronize(stream);

                unsigned long long h_count = 0;
                err = hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                if (h_count > max_batch_pairs)
                {
                    throw std::runtime_error("Batch produced more pairs than allocated: " + std::to_string(h_count));
                }

                std::vector<size_t> batch_left_indices(h_count);
                std::vector<size_t> batch_right_indices(h_count);
                if (h_count > 0)
                {
                    err = hipMemcpy(batch_left_indices.data(), d_left_idx, h_count * sizeof(size_t), hipMemcpyDeviceToHost);

                    err = hipMemcpy(batch_right_indices.data(), d_right_idx, h_count * sizeof(size_t), hipMemcpyDeviceToHost);

                    // Adjust right_indices for chunk offset
                    for (auto &idx : batch_right_indices)
                    {
                        idx += right_start;
                    }

                    left_indices.insert(left_indices.end(), batch_left_indices.begin(), batch_left_indices.end());
                    right_indices.insert(right_indices.end(), batch_right_indices.begin(), batch_right_indices.end());
                }

                // Free batch-specific allocations
                hipFree(d_left_idx);
                allocations.pop_back();
                hipFree(d_right_idx);
                allocations.pop_back();
                hipFree(d_count);
                allocations.pop_back();
                hipFree(d_right_data);
                allocations.pop_back();
            }
        }
        else if (left_col.type == DataType::DATETIME)
        {
            uint64_t *d_left_data = nullptr;
            hipError_t err = hipMalloc(&d_left_data, row_count_left * sizeof(uint64_t));

            allocations.push_back(d_left_data);

            err = hipMemcpy(d_left_data, left_table.data[index_left], row_count_left * sizeof(uint64_t), hipMemcpyHostToDevice);

            for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx)
            {
                size_t right_start = chunk_idx * chunk_size;
                size_t right_rows = std::min(chunk_size, row_count_right - right_start);
                if (right_rows == 0)
                    continue;

                hipStream_t stream = streams[chunk_idx % num_streams];

                size_t max_batch_pairs = row_count_left * right_rows;
                size_t *d_left_idx = nullptr;
                size_t *d_right_idx = nullptr;
                unsigned long long *d_count = nullptr;

                err = hipMalloc(&d_left_idx, max_batch_pairs * sizeof(size_t));

                allocations.push_back(d_left_idx);

                err = hipMalloc(&d_right_idx, max_batch_pairs * sizeof(size_t));

                allocations.push_back(d_right_idx);

                err = hipMalloc(&d_count, sizeof(unsigned long long));

                allocations.push_back(d_count);

                err = hipMemset(d_count, 0, sizeof(unsigned long long));

                uint64_t *d_right_data = nullptr;
                err = hipMalloc(&d_right_data, right_rows * sizeof(uint64_t));

                allocations.push_back(d_right_data);

                err = hipMemcpyAsync(d_right_data,
                                      static_cast<uint64_t *>(right_table.data[index_right]) + right_start,
                                      right_rows * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

                size_t chunk_blocks = (std::max(row_count_left, right_rows) + numThreads - 1) / numThreads;
                hashJoinKernel<uint64_t><<<chunk_blocks, numThreads, shared_mem_size, stream>>>(
                    d_left_data, d_right_data,
                    row_count_left, right_rows,
                    d_left_idx, d_right_idx, d_count);
                err = hipGetLastError();

                err = hipStreamSynchronize(stream);

                unsigned long long h_count = 0;
                err = hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                if (h_count > max_batch_pairs)
                {
                    throw std::runtime_error("Batch produced more pairs than allocated: " + std::to_string(h_count));
                }

                std::vector<size_t> batch_left_indices(h_count);
                std::vector<size_t> batch_right_indices(h_count);
                if (h_count > 0)
                {
                    err = hipMemcpy(batch_left_indices.data(), d_left_idx, h_count * sizeof(size_t), hipMemcpyDeviceToHost);

                    err = hipMemcpy(batch_right_indices.data(), d_right_idx, h_count * sizeof(size_t), hipMemcpyDeviceToHost);

                    for (auto &idx : batch_right_indices)
                    {
                        idx += right_start;
                    }

                    left_indices.insert(left_indices.end(), batch_left_indices.begin(), batch_left_indices.end());
                    right_indices.insert(right_indices.end(), batch_right_indices.begin(), batch_right_indices.end());
                }

                hipFree(d_left_idx);
                allocations.pop_back();
                hipFree(d_right_idx);
                allocations.pop_back();
                hipFree(d_count);
                allocations.pop_back();
                hipFree(d_right_data);
                allocations.pop_back();
            }
        }
        else if (left_col.type == DataType::STRING)
        {
            const char **d_left_data = nullptr;
            hipError_t err = hipMalloc(&d_left_data, row_count_left * sizeof(char *));

            allocations.push_back(d_left_data);

            std::vector<char *> d_strings_left(row_count_left);
            const char **host_strings_left = static_cast<const char **>(left_table.data[index_left]);
            for (size_t i = 0; i < row_count_left; ++i)
            {
                size_t len = strlen(host_strings_left[i]) + 1;
                err = hipMalloc(&d_strings_left[i], len);

                allocations.push_back(d_strings_left[i]);
                err = hipMemcpy(d_strings_left[i], host_strings_left[i], len, hipMemcpyHostToDevice);

                err = hipMemcpy(&d_left_data[i], &d_strings_left[i], sizeof(char *), hipMemcpyHostToDevice);
            }

            for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx)
            {
                size_t right_start = chunk_idx * chunk_size;
                size_t right_rows = std::min(chunk_size, row_count_right - right_start);
                if (right_rows == 0)
                    continue;

                hipStream_t stream = streams[chunk_idx % num_streams];

                size_t max_batch_pairs = row_count_left * right_rows;
                size_t *d_left_idx = nullptr;
                size_t *d_right_idx = nullptr;
                unsigned long long *d_count = nullptr;

                err = hipMalloc(&d_left_idx, max_batch_pairs * sizeof(size_t));

                allocations.push_back(d_left_idx);

                err = hipMalloc(&d_right_idx, max_batch_pairs * sizeof(size_t));

                allocations.push_back(d_right_idx);

                err = hipMalloc(&d_count, sizeof(unsigned long long));

                allocations.push_back(d_count);

                err = hipMemset(d_count, 0, sizeof(unsigned long long));

                const char **d_right_data = nullptr;
                std::vector<char *> d_strings_right(right_rows);
                err = hipMalloc(&d_right_data, right_rows * sizeof(char *));

                allocations.push_back(d_right_data);

                const char **host_strings_right = static_cast<const char **>(right_table.data[index_right]);
                for (size_t i = 0; i < right_rows; ++i)
                {
                    size_t idx = right_start + i;
                    size_t len = strlen(host_strings_right[idx]) + 1;
                    err = hipMalloc(&d_strings_right[i], len);

                    allocations.push_back(d_strings_right[i]);
                    err = hipMemcpyAsync(d_strings_right[i], host_strings_right[idx], len, hipMemcpyHostToDevice, stream);

                    err = hipMemcpyAsync(&d_right_data[i], &d_strings_right[i], sizeof(char *), hipMemcpyHostToDevice, stream);
                }

                size_t chunk_blocks = (std::max(row_count_left, right_rows) + numThreads - 1) / numThreads;
                hashJoinKernel<const char *><<<chunk_blocks, numThreads, shared_mem_size, stream>>>(
                    d_left_data, d_right_data,
                    row_count_left, right_rows,
                    d_left_idx, d_right_idx, d_count);
                err = hipGetLastError();

                err = hipStreamSynchronize(stream);

                unsigned long long h_count = 0;
                err = hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                if (h_count > max_batch_pairs)
                {
                    throw std::runtime_error("Batch produced more pairs than allocated: " + std::to_string(h_count));
                }

                std::vector<size_t> batch_left_indices(h_count);
                std::vector<size_t> batch_right_indices(h_count);
                if (h_count > 0)
                {
                    err = hipMemcpy(batch_left_indices.data(), d_left_idx, h_count * sizeof(size_t), hipMemcpyDeviceToHost);

                    err = hipMemcpy(batch_right_indices.data(), d_right_idx, h_count * sizeof(size_t), hipMemcpyDeviceToHost);

                    for (auto &idx : batch_right_indices)
                    {
                        idx += right_start;
                    }

                    left_indices.insert(left_indices.end(), batch_left_indices.begin(), batch_left_indices.end());
                    right_indices.insert(right_indices.end(), batch_right_indices.begin(), batch_right_indices.end());
                }

                hipFree(d_left_idx);
                allocations.pop_back();
                hipFree(d_right_idx);
                allocations.pop_back();
                hipFree(d_count);
                allocations.pop_back();
                hipFree(d_right_data);
                allocations.pop_back();
                for (auto ptr : d_strings_right)
                {
                    hipFree(ptr);
                    allocations.pop_back();
                }
            }
        }
        else
        {
            throw std::runtime_error("Unsupported join column type");
        }

        for (auto &stream : streams)
        {
            hipError_t err = hipStreamDestroy(stream);
        }
        for (auto ptr : allocations)
        {
            hipFree(ptr);
        }
    }
    catch (...)
    {
        for (auto ptr : allocations)
        {
            if (ptr)
                hipFree(ptr);
        }
        for (auto &stream : streams)
        {
            hipStreamDestroy(stream);
        }
        throw;
    }
}
TableResults HashJoin::executeJoin(const TableResults &left_table, const TableResults &right_table)
{
    if (left_table.row_count == 0 || right_table.row_count == 0)
    {
        TableResults result;
        result.row_count = 0;
        return result;
    }

    std::vector<size_t> left_indices;
    std::vector<size_t> right_indices;
    getIndexOfSelectedRows(left_table, right_table, left_indices, right_indices);

    TableResults result;
    result.column_count = left_table.columns.size() + right_table.columns.size();
    result.row_count = left_indices.size();
    result.has_more = left_table.has_more;
    result.columns = left_table.columns;
    result.columns.insert(result.columns.end(), right_table.columns.begin(), right_table.columns.end());
    result.data.resize(left_table.columns.size() + right_table.columns.size());

    for (size_t i = 0; i < result.columns.size(); ++i)
    {
        result.columns[i].idx = i;
    }

    size_t *d_left_idx = nullptr;
    size_t *d_right_idx = nullptr;
    std::vector<void *> allocations;
    hipError_t err = hipMalloc(&d_left_idx, result.row_count * sizeof(size_t));

    allocations.push_back(d_left_idx);

    err = hipMalloc(&d_right_idx, result.row_count * sizeof(size_t));

    allocations.push_back(d_right_idx);

    err = hipMemcpy(d_left_idx, left_indices.data(), result.row_count * sizeof(size_t), hipMemcpyHostToDevice);

    err = hipMemcpy(d_right_idx, right_indices.data(), result.row_count * sizeof(size_t), hipMemcpyHostToDevice);

    int numThreads = 256;
    int numBlocks = (result.row_count + numThreads - 1) / numThreads;
    size_t total_columns = left_table.columns.size() + right_table.columns.size();
    std::vector<hipStream_t> streams(total_columns);

    for (auto &stream : streams)
    {
        err = hipStreamCreate(&stream);
    }

    try
    {
        size_t col_idx = 0;
        // Process left table columns
        for (size_t i = 0; i < left_table.columns.size(); ++i)
        {
            hipStream_t stream = streams[col_idx];
            switch (left_table.columns[i].type)
            {
            case DataType::FLOAT:
            {
                float *d_input = nullptr;
                float *d_output = nullptr;
                err = hipMalloc(&d_input, left_table.row_count * sizeof(float));

                allocations.push_back(d_input);
                err = hipMalloc(&d_output, result.row_count * sizeof(float));

                allocations.push_back(d_output);
                err = hipMemcpyAsync(d_input, left_table.data[i], left_table.row_count * sizeof(float), hipMemcpyHostToDevice, stream);

                getRowsKernel<float><<<numBlocks, numThreads, 0, stream>>>(d_input, d_left_idx, d_output, result.row_count);
                err = hipGetLastError();

                float *h_output_data = static_cast<float *>(malloc(result.row_count * sizeof(float)));
                result.data[col_idx] = h_output_data;
                err = hipMemcpyAsync(h_output_data, d_output, result.row_count * sizeof(float), hipMemcpyDeviceToHost, stream);

                break;
            }
            case DataType::DATETIME:
            {
                uint64_t *d_input = nullptr;
                uint64_t *d_output = nullptr;
                err = hipMalloc(&d_input, left_table.row_count * sizeof(uint64_t));

                allocations.push_back(d_input);
                err = hipMalloc(&d_output, result.row_count * sizeof(uint64_t));

                allocations.push_back(d_output);
                err = hipMemcpyAsync(d_input, left_table.data[i], left_table.row_count * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

                getRowsKernel<uint64_t><<<numBlocks, numThreads, 0, stream>>>(d_input, d_left_idx, d_output, result.row_count);
                err = hipGetLastError();

                uint64_t *h_output_data = static_cast<uint64_t *>(malloc(result.row_count * sizeof(uint64_t)));
                result.data[col_idx] = h_output_data;
                err = hipMemcpyAsync(h_output_data, d_output, result.row_count * sizeof(uint64_t), hipMemcpyDeviceToHost, stream);

                break;
            }
            case DataType::STRING:
            {
                const char **h_input_strings = static_cast<const char **>(left_table.data[i]);
                const char **h_output_strings = static_cast<const char **>(malloc(result.row_count * sizeof(char *)));
                result.data[col_idx] = h_output_strings;

                const char **d_input_strings = nullptr;
                const char **d_output_strings = nullptr;
                err = hipMalloc(&d_input_strings, left_table.row_count * sizeof(char *));

                allocations.push_back(d_input_strings);
                err = hipMalloc(&d_output_strings, result.row_count * sizeof(char *));

                allocations.push_back(d_output_strings);
                err = hipMemcpyAsync(d_input_strings, h_input_strings, left_table.row_count * sizeof(char *), hipMemcpyHostToDevice, stream);

                getRowsKernel<const char *><<<numBlocks, numThreads, 0, stream>>>(d_input_strings, d_left_idx, d_output_strings, result.row_count);
                err = hipGetLastError();

                err = hipMemcpyAsync(h_output_strings, d_output_strings, result.row_count * sizeof(char *), hipMemcpyDeviceToHost, stream);

                break;
            }
            default:
                throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(left_table.columns[i].type)));
            }
            col_idx++;
        }

        // Process right table columns
        for (size_t i = 0; i < right_table.columns.size(); ++i)
        {
            hipStream_t stream = streams[col_idx];
            switch (right_table.columns[i].type)
            {
            case DataType::FLOAT:
            {
                float *d_input = nullptr;
                float *d_output = nullptr;
                err = hipMalloc(&d_input, right_table.row_count * sizeof(float));

                allocations.push_back(d_input);
                err = hipMalloc(&d_output, result.row_count * sizeof(float));

                allocations.push_back(d_output);
                err = hipMemcpyAsync(d_input, right_table.data[i], right_table.row_count * sizeof(float), hipMemcpyHostToDevice, stream);

                getRowsKernel<float><<<numBlocks, numThreads, 0, stream>>>(d_input, d_right_idx, d_output, result.row_count);
                err = hipGetLastError();

                float *h_output_data = static_cast<float *>(malloc(result.row_count * sizeof(float)));
                result.data[col_idx] = h_output_data;
                err = hipMemcpyAsync(h_output_data, d_output, result.row_count * sizeof(float), hipMemcpyDeviceToHost, stream);

                break;
            }
            case DataType::DATETIME:
            {
                uint64_t *d_input = nullptr;
                uint64_t *d_output = nullptr;
                err = hipMalloc(&d_input, right_table.row_count * sizeof(uint64_t));

                allocations.push_back(d_input);
                err = hipMalloc(&d_output, result.row_count * sizeof(uint64_t));

                allocations.push_back(d_output);
                err = hipMemcpyAsync(d_input, right_table.data[i], right_table.row_count * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

                getRowsKernel<uint64_t><<<numBlocks, numThreads, 0, stream>>>(d_input, d_right_idx, d_output, result.row_count);
                err = hipGetLastError();

                uint64_t *h_output_data = static_cast<uint64_t *>(malloc(result.row_count * sizeof(uint64_t)));
                result.data[col_idx] = h_output_data;
                err = hipMemcpyAsync(h_output_data, d_output, result.row_count * sizeof(uint64_t), hipMemcpyDeviceToHost, stream);

                break;
            }
            case DataType::STRING:
            {
                const char **h_input_strings = static_cast<const char **>(right_table.data[i]);
                const char **h_output_strings = static_cast<const char **>(malloc(result.row_count * sizeof(char *)));
                result.data[col_idx] = h_output_strings;

                const char **d_input_strings = nullptr;
                const char **d_output_strings = nullptr;
                err = hipMalloc(&d_input_strings, right_table.row_count * sizeof(char *));

                allocations.push_back(d_input_strings);
                err = hipMalloc(&d_output_strings, result.row_count * sizeof(char *));

                allocations.push_back(d_output_strings);
                err = hipMemcpyAsync(d_input_strings, h_input_strings, right_table.row_count * sizeof(char *), hipMemcpyHostToDevice, stream);

                getRowsKernel<const char *><<<numBlocks, numThreads, 0, stream>>>(d_input_strings, d_right_idx, d_output_strings, result.row_count);
                err = hipGetLastError();

                err = hipMemcpyAsync(h_output_strings, d_output_strings, result.row_count * sizeof(char *), hipMemcpyDeviceToHost, stream);

                break;
            }
            default:
                throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(right_table.columns[i].type)));
            }
            col_idx++;
        }

        for (auto &stream : streams)
        {
            hipError_t err = hipStreamSynchronize(stream);

            err = hipStreamDestroy(stream);
        }
        for (auto ptr : allocations)
        {
            hipFree(ptr);
        }
    }
    catch (...)
    {
        for (auto ptr : allocations)
        {
            if (ptr)
                hipFree(ptr);
        }
        for (auto &stream : streams)
        {
            hipStreamDestroy(stream);
        }
        throw;
    }

    return result;
}
////////////////////////////////////////////////////////////////////////////////////
void HashJoin::getIndexOfSelectedRowsCPU(const TableResults &left_table, const TableResults &right_table,
                                         std::vector<size_t> &left_indices, std::vector<size_t> &right_indices)
{
    size_t index_left = left_table.getColumnIndex(col_table_left);
    size_t index_right = right_table.getColumnIndex(col_table_right);
    ColumnInfo left_col = left_table.columns[index_left];
    ColumnInfo right_col = right_table.columns[index_right];

    if (left_col.type != right_col.type)
    {
        throw std::runtime_error("Join columns have different types");
    }

    size_t row_count_left = left_table.row_count;
    size_t row_count_right = right_table.row_count;

    if (left_col.type == DataType::FLOAT)
    {
        // Create a hash map for left table values
        std::unordered_multimap<float, size_t> hash_map;
        const float *left_data = static_cast<const float *>(left_table.data[index_left]);

        // Populate hash map with values from left table
        for (size_t i = 0; i < row_count_left; ++i)
        {
            hash_map.insert({left_data[i], i});
        }

        // Process right table and find matches
        const float *right_data = static_cast<const float *>(right_table.data[index_right]);

        for (size_t right_idx = 0; right_idx < row_count_right; ++right_idx)
        {
            float right_val = right_data[right_idx];

            auto range = hash_map.equal_range(right_val);
            for (auto it = range.first; it != range.second; ++it)
            {
                left_indices.push_back(it->second);
                right_indices.push_back(right_idx);
            }
        }
    }
    else if (left_col.type == DataType::DATETIME)
    {
        // Create a hash map for left table values
        std::unordered_multimap<uint64_t, size_t> hash_map;
        const uint64_t *left_data = static_cast<const uint64_t *>(left_table.data[index_left]);

        // Populate hash map with values from left table
        for (size_t i = 0; i < row_count_left; ++i)
        {
            hash_map.insert({left_data[i], i});
        }

        // Process right table and find matches
        const uint64_t *right_data = static_cast<const uint64_t *>(right_table.data[index_right]);

        for (size_t right_idx = 0; right_idx < row_count_right; ++right_idx)
        {
            uint64_t right_val = right_data[right_idx];

            auto range = hash_map.equal_range(right_val);
            for (auto it = range.first; it != range.second; ++it)
            {
                left_indices.push_back(it->second);
                right_indices.push_back(right_idx);
            }
        }
    }
    else if (left_col.type == DataType::STRING)
    {
        // Create a hash map for left table values
        std::unordered_multimap<std::string, size_t> hash_map;
        const char **left_data = static_cast<const char **>(left_table.data[index_left]);

        // Populate hash map with values from left table
        for (size_t i = 0; i < row_count_left; ++i)
        {
            hash_map.insert({left_data[i], i});
        }

        // Process right table and find matches
        const char **right_data = static_cast<const char **>(right_table.data[index_right]);

        for (size_t right_idx = 0; right_idx < row_count_right; ++right_idx)
        {
            std::string right_val = right_data[right_idx];

            auto range = hash_map.equal_range(right_val);
            for (auto it = range.first; it != range.second; ++it)
            {
                left_indices.push_back(it->second);
                right_indices.push_back(right_idx);
            }
        }
    }
    else
    {
        throw std::runtime_error("Unsupported join column type");
    }
}

TableResults HashJoin::executeJoinCPU(const TableResults &left_table, const TableResults &right_table)
{
    if (left_table.row_count == 0 || right_table.row_count == 0)
    {
        TableResults result;
        result.row_count = 0;
        result.has_more = false;
        return result;
    }

    std::vector<size_t> left_indices;
    std::vector<size_t> right_indices;
    getIndexOfSelectedRowsCPU(left_table, right_table, left_indices, right_indices);

    TableResults result;
    result.column_count = left_table.columns.size() + right_table.columns.size();
    result.row_count = left_indices.size();
    result.has_more = left_table.has_more;
    result.columns = left_table.columns;
    result.columns.insert(result.columns.end(), right_table.columns.begin(), right_table.columns.end());
    result.data.resize(result.column_count, nullptr);

    for (size_t i = 0; i < result.columns.size(); ++i)
    {
        result.columns[i].idx = i;
    }

    // Process left table columns
    for (size_t i = 0; i < left_table.columns.size(); ++i)
    {
        switch (left_table.columns[i].type)
        {
        case DataType::FLOAT:
        {
            const float *input_data = static_cast<const float *>(left_table.data[i]);
            float *output_data = static_cast<float *>(malloc(result.row_count * sizeof(float)));
            result.data[i] = output_data;

            // Copy data from left table to result using indices
            for (size_t j = 0; j < result.row_count; ++j)
            {
                output_data[j] = input_data[left_indices[j]];
            }
            break;
        }
        case DataType::DATETIME:
        {
            const uint64_t *input_data = static_cast<const uint64_t *>(left_table.data[i]);
            uint64_t *output_data = static_cast<uint64_t *>(malloc(result.row_count * sizeof(uint64_t)));
            result.data[i] = output_data;

            for (size_t j = 0; j < result.row_count; ++j)
            {
                output_data[j] = input_data[left_indices[j]];
            }
            break;
        }
        case DataType::STRING:
        {
            const char **input_strings = static_cast<const char **>(left_table.data[i]);
            const char **output_strings = static_cast<const char **>(malloc(result.row_count * sizeof(char *)));
            result.data[i] = output_strings;

            for (size_t j = 0; j < result.row_count; ++j)
            {
                output_strings[j] = input_strings[left_indices[j]];
            }
            break;
        }
        default:
            throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(left_table.columns[i].type)));
        }
    }

    // Process right table columns
    for (size_t i = 0; i < right_table.columns.size(); ++i)
    {
        size_t result_col_idx = left_table.columns.size() + i;

        switch (right_table.columns[i].type)
        {
        case DataType::FLOAT:
        {
            const float *input_data = static_cast<const float *>(right_table.data[i]);
            float *output_data = static_cast<float *>(malloc(result.row_count * sizeof(float)));
            result.data[result_col_idx] = output_data;

            for (size_t j = 0; j < result.row_count; ++j)
            {
                output_data[j] = input_data[right_indices[j]];
            }
            break;
        }
        case DataType::DATETIME:
        {
            const uint64_t *input_data = static_cast<const uint64_t *>(right_table.data[i]);
            uint64_t *output_data = static_cast<uint64_t *>(malloc(result.row_count * sizeof(uint64_t)));
            result.data[result_col_idx] = output_data;

            for (size_t j = 0; j < result.row_count; ++j)
            {
                output_data[j] = input_data[right_indices[j]];
            }
            break;
        }
        case DataType::STRING:
        {
            const char **input_strings = static_cast<const char **>(right_table.data[i]);
            const char **output_strings = static_cast<const char **>(malloc(result.row_count * sizeof(char *)));
            result.data[result_col_idx] = output_strings;

            for (size_t j = 0; j < result.row_count; ++j)
            {
                output_strings[j] = input_strings[right_indices[j]];
            }
            break;
        }
        default:
            throw std::runtime_error("Unsupported data type: " + std::to_string(static_cast<int>(right_table.columns[i].type)));
        }
    }

    return result;
}
void HashJoin::print() const
{
    std::cout << "HashJoin: " << col_table_left << " = " << col_table_right << std::endl;
    // for (const auto &child : children)
    // {
    //     child->print(os, indent + 2);
    // }
}
